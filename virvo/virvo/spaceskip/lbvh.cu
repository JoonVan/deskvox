#include "hip/hip_runtime.h"
// Virvo - Virtual Reality Volume Rendering
// Copyright (C) 1999-2003 University of Stuttgart, 2004-2005 Brown University
// Contact: Jurgen P. Schulze, jschulze@ucsd.edu
//
// This file is part of Virvo.
//
// Virvo is free software; you can redistribute it and/or
// modify it under the terms of the GNU Lesser General Public
// License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
//
// This library is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
// Lesser General Public License for more details.
//
// You should have received a copy of the GNU Lesser General Public
// License along with this library (see license.txt); if not, write to the
// Free Software Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA

#ifndef __CUDACC_EXTENDED_LAMBDA__
#error "Compile w/ option --expt-extended-lambda"
#endif

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

#undef MATH_NAMESPACE
#include <visionaray/math/detail/math.h> // div_up
#include <visionaray/math/aabb.h>
#include <visionaray/morton.h>
#undef MATH_NAMESPACE

#include "../cuda/timer.h"
#include "../vvopengl.h"
#include "../vvspaceskip.h"
#include "../vvvoldesc.h"
#include "lbvh.h"

using namespace visionaray;


//-------------------------------------------------------------------------------------------------
// Compact brick data structure. max-corner is explicitly given by adding global brick-size
// to min-corner!
//

struct VSNRAY_ALIGN(16) Brick
{
  int min_corner[3];
  // After compaction, all bricks are non-empty
  // anyway ==> reuse this field afterwards
  union
  {
    unsigned morton_code;
    int is_empty = true;
  };
};


//-------------------------------------------------------------------------------------------------
// Tree node that can be stored in device memory
//

struct Node
{
  __device__ Node()
    : bbox(vec3i(INT_MAX), vec3i(0))
  {
  }

  aabbi bbox;
  int left = -1;
  int right = -1;
  int parent = -1;
};


//-------------------------------------------------------------------------------------------------
// Helpers
//

template <typename T>
__device__
inline int signum(T a)
{
    return (T(0.0) < a) - (a < T(0.0));
}


//-------------------------------------------------------------------------------------------------
// Find node range that an inner node overlaps
//

__device__
vec2i determine_range(Brick* bricks, int num_bricks, int i)
{
  auto delta = [&](int i, int j)
  {
    // Karras' delta(i,j) function
    // Denotes the length of the longest common
    // prefix between keys k_i and k_j

    // Cf. Figure 4: "for simplicity, we define that
    // delta(i,j) = -1 when j not in [0,n-1]"
    if (j < 0 || j >= num_bricks)
      return -1;

    return __clz(bricks[i].morton_code ^ bricks[j].morton_code);
  };

  int num_inner = num_bricks - 1;

  if (i == 0)
    return { 0, num_inner };

  // Determine direction of the range (+1 or -1)
  int d = signum(delta(i, i + 1) - delta(i, i - 1));

  // Compute upper bound for the length of the range
  int delta_min = delta(i, i - d);
  int l_max = 2;
  while (delta(i, i + l_max * d) > delta_min)
  {
    l_max *= 2;
  }

  // Find the other end using binary search
  int l = 0;
  for (int t = l_max >> 1; t >= 1; t >>= 1)
  {
    if (delta(i, i + (l + t) * d) > delta_min)
      l += t;
  }

  if (d == 1)
    return vec2i(i, i + l * d);
  else
    return vec2i(i + l * d, i);
}


//-------------------------------------------------------------------------------------------------
// Find split positions based on Morton codes
//

__device__
int find_split(Brick* bricks, int first, int last)
{
  unsigned code_first = bricks[first].morton_code;
  unsigned code_last  = bricks[last].morton_code;

  if (code_first == code_last)
  {
    return (first + last) / 2;
  }

  unsigned common_prefix = __clz(code_first ^ code_last);

  int result = first;
  int step = last - first;

  do
  {
    step = (step + 1) / 2;
    int next = result + step;

    if (next < last)
    {
      unsigned code = bricks[next].morton_code;
      if (code_first == code || __clz(code_first ^ code) > common_prefix)
      {
        result = next;
      }
    }
  }
  while (step > 1);

  return result;
}


//-------------------------------------------------------------------------------------------------
// Kernels
//

template <typename TransfuncTex>
__global__ void findNonEmptyBricks(const float* voxels, TransfuncTex transfunc, Brick* bricks)
{
  unsigned brick_index = blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x;
  unsigned brick_offset = brick_index * blockDim.x * blockDim.y * blockDim.z;

  unsigned index = brick_offset + threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

  __shared__ int shared_empty;
  shared_empty = true;

  __syncthreads();

  bool empty = tex1D(transfunc, voxels[index]).w < 0.0001f;
  // All threads in block vote
  if (shared_empty && !empty)
    atomicExch(&shared_empty, false);

  __syncthreads();

  if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
  {
    bricks[brick_index].min_corner[0] = blockIdx.x;
    bricks[brick_index].min_corner[1] = blockIdx.y;
    bricks[brick_index].min_corner[2] = blockIdx.z;

    if (!shared_empty)
      bricks[brick_index].is_empty = false;
  }
}

__global__ void assignMortonCodes(Brick* bricks, int num_bricks)
{
  unsigned index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < num_bricks)
  {
    Brick& b = bricks[index];
    b.morton_code = morton_encode3D(b.min_corner[0], b.min_corner[1], b.min_corner[2]);
  }
}

__global__ void nodeSplitting(Brick* bricks, int num_bricks, Node* leaves, Node* inner)
{
  int num_leaves = num_bricks;
  int num_inner = num_leaves - 1;

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < num_inner)
  {
    // NOTE: This is [first..last], not [first..last)!!
    vec2i range = determine_range(bricks, num_bricks, index);
    int first = range.x;
    int last = range.y;

    int split = find_split(bricks, first, last);
    //printf("%d: %d %d %d\n", index, first, split, last);

    int left = split;
    int right = split + 1;

    if (left == first)
    {
      // left child is leaf
      inner[index].left = ~left;
      leaves[left].parent = index;
    }
    else
    {
      // left child is inner
      inner[index].left = left;
      inner[left].parent = index;
    }

    if (right == last)
    {
      // right child is leaf
      inner[index].right = ~right;
      leaves[right].parent = index;
    }
    else
    {
      // right child is inner
      inner[index].right = right;
      inner[right].parent = index;
    }
  }
}

__global__ void buildHierarchy(Node* inner,
        int num_inner,
        Node* leaves,
        int num_leaves,
        Brick* bricks,
        virvo::SkipTreeNode* nodes,
        vec3i vox,
        vec3 dist,
        float scale)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= num_leaves)
    return;

  // Leaf's bounding box
  aabbi bbox(vec3i(bricks[index].min_corner)*8, vec3i(bricks[index].min_corner)*8 + vec3i(8,8,8));
  leaves[index].bbox = bbox;

  // Atomically combine child bounding boxes and update parents
  int next = leaves[index].parent;

  while (next >= 0)
  {
    atomicMin(&inner[next].bbox.min.x, bbox.min.x);
    atomicMin(&inner[next].bbox.min.y, bbox.min.y);
    atomicMin(&inner[next].bbox.min.z, bbox.min.z);
    atomicMax(&inner[next].bbox.max.x, bbox.max.x);
    atomicMax(&inner[next].bbox.max.y, bbox.max.y);
    atomicMax(&inner[next].bbox.max.z, bbox.max.z);
    next = inner[next].parent;
  }

  __threadfence();

  // Convert aabbi to aabb. Each thread (but one) processes an inner node and a leaf
  // Also set indices while we're at it!

  if (index < num_inner)
  {
    auto bbox = inner[index].bbox;
    bbox.min.y = vox[1] - inner[index].bbox.max.y;
    bbox.max.y = vox[1] - inner[index].bbox.min.y;
    bbox.min.z = vox[2] - inner[index].bbox.max.z;
    bbox.max.z = vox[2] - inner[index].bbox.min.z;
    vec3 bmin = (vec3(bbox.min) - vec3(vox)/2.f) * dist * scale;
    vec3 bmax = (vec3(bbox.max) - vec3(vox)/2.f) * dist * scale;
    nodes[index].min_corner[0] = bmin.x;
    nodes[index].min_corner[1] = bmin.y;
    nodes[index].min_corner[2] = bmin.z;
    nodes[index].left = inner[index].left >= 0 ? inner[index].left : ~(num_inner + ~inner[index].left);
    nodes[index].max_corner[0] = bmax.x;
    nodes[index].max_corner[1] = bmax.y;
    nodes[index].max_corner[2] = bmax.z;
    nodes[index].right = inner[index].right >= 0 ? inner[index].right : ~(num_inner + ~inner[index].right);
  }

//if (index < num_leaves)
  {
    auto bbox = leaves[index].bbox;
    bbox.min.y = vox[1] - leaves[index].bbox.max.y;
    bbox.max.y = vox[1] - leaves[index].bbox.min.y;
    bbox.min.z = vox[2] - leaves[index].bbox.max.z;
    bbox.max.z = vox[2] - leaves[index].bbox.min.z;
    vec3 bmin = (vec3(bbox.min) - vec3(vox)/2.f) * dist * scale;
    vec3 bmax = (vec3(bbox.max) - vec3(vox)/2.f) * dist * scale;
    nodes[num_inner + index].min_corner[0] = bmin.x;
    nodes[num_inner + index].min_corner[1] = bmin.y;
    nodes[num_inner + index].min_corner[2] = bmin.z;
    nodes[num_inner + index].left = -1;
    nodes[num_inner + index].max_corner[0] = bmax.x;
    nodes[num_inner + index].max_corner[1] = bmax.y;
    nodes[num_inner + index].max_corner[2] = bmax.z;
    nodes[num_inner + index].right = -1;
  }
}


//-------------------------------------------------------------------------------------------------
// BVH private implementation
//

struct BVH::Impl
{
  vec3i vox;
  vec3 dist;
  float scale;
  // Brickwise (8x8x8) sorted on a z-order curve, "natural" layout inside!
  thrust::device_vector<float> voxels;
  thrust::device_vector<virvo::SkipTreeNode> nodes;
};


//-------------------------------------------------------------------------------------------------
// BVH
//

BVH::BVH()
  : impl_(new Impl)
{
}

BVH::~BVH()
{
}

void BVH::updateVolume(vvVolDesc const& vd, int channel)
{
  impl_->vox = vec3i(vd.vox.x, vd.vox.y, vd.vox.z);
  impl_->dist = vec3(vd.getDist().x, vd.getDist().y, vd.getDist().z);
  impl_->scale = vd._scale;

  vec3i brick_size(8,8,8);

  vec3i num_bricks(div_up(impl_->vox[0], brick_size.x),
      div_up(impl_->vox[1], brick_size.y),
      div_up(impl_->vox[2], brick_size.z));

  size_t num_voxels = num_bricks.x*brick_size.x * num_bricks.y*brick_size.y * num_bricks.z*brick_size.z;

  thrust::host_vector<float> host_voxels(num_voxels);

  for (int bz = 0; bz < num_bricks.z; ++bz)
  {
    for (int by = 0; by < num_bricks.y; ++by)
    {
      for (int bx = 0; bx < num_bricks.x; ++bx)
      {
        // Brick index
        int brick_index = bz * num_bricks.x * num_bricks.y + by * num_bricks.x + bx;
        // Brick offset in voxels array
        int brick_offset = brick_index * brick_size.x * brick_size.y * brick_size.z;

        for (int zz = 0; zz < brick_size.z; ++zz)
        {
          for (int yy = 0; yy < brick_size.y; ++yy)
          {
            for (int xx = 0; xx < brick_size.x; ++xx)
            {
              // Index into voxels array
              int index = brick_offset + zz * brick_size.x * brick_size.y + yy * brick_size.x + xx;

              // Indices into voldesc
              int x = bx * brick_size.x + xx;
              int y = by * brick_size.y + yy;
              int z = bz * brick_size.z + zz;

              if (x < impl_->vox[0] && y < impl_->vox[1] && z < impl_->vox[2])
              {
                host_voxels[index] = vd.getChannelValue(vd.getCurrentFrame(),
                    x,
                    y,
                    z,
                    channel);
              }
              else
                host_voxels[index] = 0.f;
            }
          }
        }
      }
    }
  }

  impl_->voxels.resize(host_voxels.size());
  thrust::copy(host_voxels.begin(), host_voxels.end(), impl_->voxels.begin());
}

void BVH::updateTransfunc(BVH::TransfuncTex transfunc)
{
  cuda_texture<visionaray::vec4, 1> cuda_transfunc(transfunc.data(),
      transfunc.width(),
      transfunc.get_address_mode(),
      transfunc.get_filter_mode());

  dim3 block_size(8, 8, 8);
  dim3 grid_size(div_up(impl_->vox[0], (int)block_size.x),
                 div_up(impl_->vox[1], (int)block_size.y),
                 div_up(impl_->vox[2], (int)block_size.z));

  // Identify non-empty bricks
  thrust::device_vector<Brick> bricks(grid_size.x * grid_size.y * grid_size.z);

  virvo::CudaTimer t;
  findNonEmptyBricks<<<grid_size, block_size>>>(
      thrust::raw_pointer_cast(impl_->voxels.data()),
      cuda_texture_ref<visionaray::vec4, 1>(cuda_transfunc),
      thrust::raw_pointer_cast(bricks.data()));
  std::cout << "Find empty: " << t.elapsed() << '\n';
  t.reset();

  // Compact non-empty bricks to the left of the list
  thrust::device_vector<Brick> compact_bricks(grid_size.x * grid_size.y * grid_size.z);

  auto last = thrust::copy_if(
      thrust::device,
      bricks.begin(),
      bricks.end(),
      compact_bricks.begin(),
      [] __device__ (Brick b) { return !b.is_empty; });
  std::cout << "Compaction: " << t.elapsed() << '\n';
  t.reset();

  size_t numNonEmptyBricks = last - compact_bricks.begin();
  size_t numThreads = 1024;

  assignMortonCodes<<<div_up(numNonEmptyBricks, numThreads), numThreads>>>(
      thrust::raw_pointer_cast(compact_bricks.data()),
      numNonEmptyBricks);
  std::cout << "Assign Morton: " << t.elapsed() << '\n';
  t.reset();

  thrust::stable_sort(
      thrust::device,
      compact_bricks.begin(),
      last,
      [] __device__ (Brick l, Brick r)
      {
        return l.morton_code < r.morton_code;
      });
  std::cout << "Sorting: " << t.elapsed() << '\n';
  t.reset();

#if 0
  thrust::host_vector<Brick> h_karras_bricks(8);
  h_karras_bricks[0].morton_code = 1;
  h_karras_bricks[1].morton_code = 2;
  h_karras_bricks[2].morton_code = 4;
  h_karras_bricks[3].morton_code = 5;
  h_karras_bricks[4].morton_code = 19;
  h_karras_bricks[5].morton_code = 24;
  h_karras_bricks[6].morton_code = 25;
  h_karras_bricks[7].morton_code = 30;
  thrust::device_vector<Brick> karras_bricks(h_karras_bricks);
  thrust::device_vector<Node> leaves(8);
  thrust::device_vector<Node> inner(7);
  nodeSplitting<<<div_up(numNonEmptyBricks, numThreads), numThreads>>>(
        thrust::raw_pointer_cast(karras_bricks.data()),
        8,
        thrust::raw_pointer_cast(leaves.data()),
        thrust::raw_pointer_cast(inner.data()));
#else
  thrust::device_vector<Node> leaves(numNonEmptyBricks);
  thrust::device_vector<Node> inner(numNonEmptyBricks - 1);
  nodeSplitting<<<div_up(numNonEmptyBricks, numThreads), numThreads>>>(
      thrust::raw_pointer_cast(compact_bricks.data()),
      numNonEmptyBricks,
      thrust::raw_pointer_cast(leaves.data()),
      thrust::raw_pointer_cast(inner.data()));
  std::cout << "Splitting: " << t.elapsed() << '\n';
  t.reset();

#endif
#if 0
  thrust::host_vector<Node> h_inner(inner);
  int i = 0;
  for (auto n : h_inner)
  {
    auto l = n.left >= 0 ? n.left : ~n.left;
    auto r = n.right >= 0 ? n.right : ~n.right;
    auto strl = n.left >= 0 ? "INNER" : "LEAF";
    auto strr = n.right >= 0 ? "INNER" : "LEAF";
    std::cout << i++ << ": "
              << "Left: " << strl << '(' << l << "), "
              << "right: " << strr << '(' << r << "), "
              << "parent: " << n.parent << '\n';
  }
#endif

  virvo::SkipTreeNode init = { FLT_MAX, FLT_MAX, FLT_MAX, -1, -FLT_MAX, -FLT_MAX, -FLT_MAX, -1 };
  impl_->nodes.resize(inner.size() + leaves.size(), init);

  buildHierarchy<<<div_up(leaves.size(), numThreads), numThreads>>>(
      thrust::raw_pointer_cast(inner.data()),
      inner.size(),
      thrust::raw_pointer_cast(leaves.data()),
      leaves.size(),
      thrust::raw_pointer_cast(compact_bricks.data()),
      thrust::raw_pointer_cast(impl_->nodes.data()),
      impl_->vox,
      impl_->dist,
      impl_->scale);
  std::cout << "Build hierarchy: " << t.elapsed() << '\n';
}

virvo::SkipTreeNode* BVH::getNodes(int& numNodes)
{
  numNodes = impl_->nodes.size();
  return thrust::raw_pointer_cast(impl_->nodes.data());
}

std::vector<aabb> BVH::get_leaf_nodes(vec3 eye, bool frontToBack) const
{
  // TODO: it should also be possible to directly return
  // a device pointer to the leaf nodes

  // There are n-1 inner nodes followed by n leaves
  int num_inner = impl_->nodes.size() / 2;
  int num_leaves = impl_->nodes.size() - num_inner;

  std::vector<virvo::SkipTreeNode> leaves(num_leaves);
  thrust::copy(
      impl_->nodes.data() + num_inner,
      impl_->nodes.data() + num_inner + num_leaves,
      leaves.data());

  std::vector<aabb> result(num_leaves);

  for (size_t i = 0; i < leaves.size(); ++i)
  {
    result[i].min = vec3(leaves[i].min_corner);
    result[i].max = vec3(leaves[i].max_corner);
  }

  std::sort(
      result.begin(),
      result.end(),
      [eye, frontToBack](aabb const& l, aabb const& r)
      {
        auto distl = length(eye - l.center());
        auto distr = length(eye - r.center());

        if (frontToBack)
          return distl < distr;
        else
          return distr < distl;
      });

  return result;
}

void BVH::renderGL(vvColor color) const
{
  int numNodes = 0;
  auto nodes = const_cast<BVH*>(this)->getNodes(numNodes); // TODO..

  std::vector<virvo::SkipTreeNode> h_nodes(numNodes);
  hipMemcpy(h_nodes.data(),
      nodes,
      numNodes * sizeof(virvo::SkipTreeNode),
      hipMemcpyDeviceToHost);

  for (auto n : h_nodes)
  {
    vec3 bmin(n.min_corner);
    vec3 bmax(n.max_corner);

    glBegin(GL_LINES);
    glColor3f(color[0], color[1], color[2]);

    glVertex3f(bmin.x, bmin.y, bmin.z);
    glVertex3f(bmax.x, bmin.y, bmin.z);

    glVertex3f(bmax.x, bmin.y, bmin.z);
    glVertex3f(bmax.x, bmax.y, bmin.z);

    glVertex3f(bmax.x, bmax.y, bmin.z);
    glVertex3f(bmin.x, bmax.y, bmin.z);

    glVertex3f(bmin.x, bmax.y, bmin.z);
    glVertex3f(bmin.x, bmin.y, bmin.z);

    //
    glVertex3f(bmin.x, bmin.y, bmax.z);
    glVertex3f(bmax.x, bmin.y, bmax.z);

    glVertex3f(bmax.x, bmin.y, bmax.z);
    glVertex3f(bmax.x, bmax.y, bmax.z);

    glVertex3f(bmax.x, bmax.y, bmax.z);
    glVertex3f(bmin.x, bmax.y, bmax.z);

    glVertex3f(bmin.x, bmax.y, bmax.z);
    glVertex3f(bmin.x, bmin.y, bmax.z);

    //
    glVertex3f(bmin.x, bmin.y, bmin.z);
    glVertex3f(bmin.x, bmin.y, bmax.z);

    glVertex3f(bmax.x, bmin.y, bmin.z);
    glVertex3f(bmax.x, bmin.y, bmax.z);

    glVertex3f(bmax.x, bmax.y, bmin.z);
    glVertex3f(bmax.x, bmax.y, bmax.z);

    glVertex3f(bmin.x, bmax.y, bmin.z);
    glVertex3f(bmin.x, bmax.y, bmax.z);
    glEnd();
  }
}

//============================================================================
// End of File
//============================================================================
// vim: sw=2:expandtab:softtabstop=2:ts=2:cino=\:0g0t0
